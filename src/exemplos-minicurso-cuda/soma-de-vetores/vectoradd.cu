#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

// Host input vectors.
float *h_a;
float *h_b;
// Host output vector.
float *h_c;

// Device input vectors.
float *d_a;
float *d_b;
// Device output vector.
float *d_c;

// Size of arrays.
int n = 0;

/* CUDA kernel. Each thread takes care of one element of c. */
__global__ void vecAdd(float *a, float *b, float *c, int n) {
	// Get our global thread ID
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	// Make sure we do not go out of bounds
	if (id < n)
		c[id] = a[id] + b[id];
}

void init_array() {
	fprintf(stdout, "Inicializando os arrays.\n");
	int i;
	// Initialize vectors on host.
        for (i = 0; i < n; i++) {
		h_a[i] = sinf(i) * sinf(i);
		h_b[i] = cosf(i) * cosf(i);
        }
}

void print_array() {
	int i;
	printf("Imprimindo o Resultado.\n");
	for (i = 0; i < n; i++) {
		fprintf(stdout, "h_c[%07d]: %f\n", i, h_c[i]);
  	}
}

void check_result(){
	// Soma dos elementos do array C e divide por N, o valor deve ser igual a 1.
	int i;
	float sum = 0;
	fprintf(stdout, "Verificando o Resultado.\n");  
	for (i = 0; i < n; i++) {
		sum += h_c[i];
	}
	
	fprintf(stdout, "Resultado Final: (%f, %f)\n", sum, (float)(sum / (float)n));
}

/* Main code */
int main(int argc, char *argv[]) {
	// Size of vectors
	n = atoi(argv[1]);

	printf("Número de Elementos: %d\n", n);

	// Size, in bytes, of each vector
	size_t bytes = n * sizeof(float);
	printf("Memória que será alocada para os 3 arrays: %d\n", 3 * bytes);

	printf("Allocate memory for each vector on host\n");
	// Allocate memory for each vector on host
	h_a = (float *)malloc(bytes);
	h_b = (float *)malloc(bytes);
	h_c = (float *)malloc(bytes);

	printf("Allocate memory for each vector on GPU\n");
	// Allocate memory for each vector on GPU
	hipMalloc(&d_a, bytes);
	hipMalloc(&d_b, bytes);
	hipMalloc(&d_c, bytes);

	printf("Initialize vectors on host\n");
	init_array();

	printf("Copy host vectors to device\n");
	// Copy host vectors to device
	hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

	// Number of threads in each thread block.
	int threadsPerBlock = 256;
	// Number of thread blocks in grid.
	int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

	printf("Execute the kernel\n");
	hipEvent_t start_event, stop_event;
	float time_kernel_execution;
	int eventflags = hipEventBlockingSync;
	hipEventCreateWithFlags(&start_event, eventflags);
	hipEventCreateWithFlags(&stop_event, eventflags);

	/* Recording the time to kernel execution */
	hipEventRecord(start_event, 0);

	/* Execute the kernel. */
	vecAdd <<< blocksPerGrid, threadsPerBlock >>> (d_a, d_b, d_c, n);

	/* Synchronize */
	hipDeviceSynchronize();

	hipEventRecord(stop_event, 0);
	hipEventSynchronize(stop_event);
	hipEventElapsedTime(&time_kernel_execution, start_event, stop_event);
	printf("Time Kernel Execution: %f s\n", (time_kernel_execution / 1000.0f));

	printf("Copy array back to host\n");
	// Copy array back to host
	hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

	print_array();

	check_result();

	printf("Time Kernel Execution: %f ms\n", (time_kernel_execution));

	// Release device memory
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	// Release host memory
	free(h_a);
	free(h_b);
	free(h_c);

	return 0;
}
