#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

// Host input vectors.
float *uva_a;
float *uva_b;
// Host output vector.
float *uva_c;

// Size of arrays.
int n = 0;

/* CUDA kernel. Each thread takes care of one element of c. */
__global__ void vecAdd(float *a, float *b, float *c, int n) {
	// Get our global thread ID
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	// Make sure we do not go out of bounds
	if (id < n)
		c[id] = a[id] + b[id];
}

void init_array() {
	fprintf(stdout, "Inicializando os arrays.\n");
	int i;
	// Initialize vectors on host.
        for (i = 0; i < n; i++) {
		uva_a[i] = sinf(i) * sinf(i);
		uva_b[i] = cosf(i) * cosf(i);
        }
}

void print_array() {
	int i;
	printf("Imprimindo o Resultado.\n");
	for (i = 0; i < n; i++) {
		fprintf(stdout, "uva_c[%07d]: %f\n", i, uva_c[i]);
  	}
}

void check_result(){
	// Soma dos elementos do array C e divide por N, o valor deve ser igual a 1.
	int i;
	float sum = 0;
	fprintf(stdout, "Verificando o Resultado.\n");  
	for (i = 0; i < n; i++) {
		sum += uva_c[i];
	}
	
	fprintf(stdout, "Resultado Final: (%f, %f)\n", sum, (float)(sum / (float)n));
}

/* Main code */
int main(int argc, char *argv[]) {
	// Size of vectors
	n = atoi(argv[1]);

	printf("Número de Elementos: %d\n", n);

	// Size, in bytes, of each vector
	size_t bytes = n * sizeof(float);
	printf("Memória que será alocada para os 3 arrays: %d\n", 3 * bytes);

	printf("Allocate memory for each vector on host\n");
	// Allocate memory for each vector on host
	hipMallocManaged(&uva_a, bytes);
	hipMallocManaged(&uva_b, bytes);
	hipMallocManaged(&uva_c, bytes);

	printf("Initialize vectors on host\n");
	init_array();

	// Number of threads in each thread block.
	int threadsPerBlock = 256;
	// Number of thread blocks in grid.
	int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

	printf("Execute the kernel\n");
	hipEvent_t start_event, stop_event;
	float time_kernel_execution;
	int eventflags = hipEventBlockingSync;
	hipEventCreateWithFlags(&start_event, eventflags);
	hipEventCreateWithFlags(&stop_event, eventflags);

	/* Recording the time to kernel execution */
	hipEventRecord(start_event, 0);

	/* Execute the kernel. */
	vecAdd <<< blocksPerGrid, threadsPerBlock >>> (uva_a, uva_b, uva_c, n);

	/* Synchronize */
	hipDeviceSynchronize();

	hipEventRecord(stop_event, 0);
	hipEventSynchronize(stop_event);
	hipEventElapsedTime(&time_kernel_execution, start_event, stop_event);
	printf("Time Kernel Execution: %f s\n", (time_kernel_execution / 1000.0f));

	print_array();

	check_result();

	printf("Time Kernel Execution: %f ms\n", (time_kernel_execution));

	// Release device memory
	hipFree(uva_a);
	hipFree(uva_b);
	hipFree(uva_c);

	return 0;
}
