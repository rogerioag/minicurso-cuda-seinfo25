#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "dimensions.h"


__global__ void checkIndex(int funcId) {
  /*printf("threadIdx:(%2d, %2d, %2d) blockIdx:(%2d, %2d, %2d) blockDim:(%2d, %2d, %2d) "
         "gridDim:(%2d, %2d, %2d) -> id: %2d\n",
         threadIdx.x, threadIdx.y, threadIdx.z, blockIdx.x, blockIdx.y,
         blockIdx.z, blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y,
         gridDim.z, getGlobalIdFunc[funcId]());*/
    printf("gridDim:(%2d, %2d, %2d) blockDim:(%2d, %2d, %2d) blockIdx:(%2d, %2d, %2d) "
         "threadIdx:(%2d, %2d, %2d) -> id: %2d\n", gridDim.x, gridDim.y,
         gridDim.z, blockDim.x, blockDim.y, blockDim.z, blockIdx.x, blockIdx.y,
         blockIdx.z, threadIdx.x, threadIdx.y, threadIdx.z, getGlobalIdFunc[funcId]());
    
}

int main(int argc, char **argv) {

  if (argc != 8) {
        printf("Uso: %s <g.x> <g.y> <g.z> <b.x> <b.y> <b.z> <gpuId>\n", argv[0]);
        return 0;
    }
  /* Definição do arranjo de threads em blocos do grid. */
  int gx = atoi(argv[1]);
  int gy = atoi(argv[2]);
  int gz = atoi(argv[3]);
  int bx = atoi(argv[4]);
  int by = atoi(argv[5]);
  int bz = atoi(argv[6]);

  dim3 grid(gx, gy, gz);
  dim3 block(bx, by, bz);

  printf("config(gx: %d, gy: %d, gz: %d, bx: %d, by: %d, bz: %d)\n", grid.x, grid.y, grid.z, block.x, block.y, block.z);

  /*
     grid(gx,gy,gx) block(bx,by,bz)
     funcId é escolhida com base nos valores de [gx,gy,gx,bx,by,bz]
     Cada valor irá contribuir com uma parcela para o cálculo do índice da função:
     [gx > 1, gy > 1, gx > 1, bx > 1, by > 1, bz > 1]
     Exemplo: grid(32,1,1) block(32,1,1)
              [1,0,0,1,0,0] -> [32,16,8,4,2,1] = [32 + 4] = 36
              A função getGlobalIdFunc(36) será:
              // 36: 100 100 getGlobalIdx_grid_1D_x_block_1D_x 
  */

  int funcId = calculateFunctionId(grid, block);

  printf("funcId: %d\n", funcId);

  int gpuId =  atoi(argv[7]);

  /* Define the gpu id to work */
  hipSetDevice(gpuId);

  // check grid and block dimension from host side
  printf("config(gx: %d, gy: %d, gz: %d, bx: %d, by: %d, bz: %d)\n", grid.x, grid.y, grid.z, block.x, block.y, block.z);
  
  printf("gridDim:( x,  y,  z) blockDim:( x,  y,  z) blockIdx:( x,  y,  z) threadIdx:( x,  y,  z)\n");
  // check grid and block dimension from device side
  checkIndex<<<grid, block>>>(funcId);
  
  // reset device before you leave
  hipDeviceReset();
  return (0);
}
